#include "hip/hip_runtime.h"
#include <assert.h>
#include <cstdlib>
#include <iostream>
#include <hiprand.h>
#include <hipblas.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include "cublas_functions.h"
#include "matrix_utils.h"

/* GPU kernels */

/* Convert double into float for all matrix values */
template <int BLOCK_SIZE> __global__ void fillFloatMatrices(double *D, float *F){
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    F[threadId] = (float) D[threadId];
}



/* CPU functions */
void gpu_blas_cgemm(hipblasHandle_t handle, hipFloatComplex *A, hipFloatComplex *B, 
        hipFloatComplex *C, const int n){
    int lda=n,ldb=n,ldc=n;
    const hipFloatComplex alf = {(float)1., (float)1.};
    const hipFloatComplex bet = {0.,0.};
    const hipFloatComplex *alpha = &alf;
    const hipFloatComplex *beta = &bet;
    hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, alpha, 
            A, lda, 
            B, ldb, beta,
            C, ldc);
}
void gpu_blas_zgemm(hipblasHandle_t handle,  hipDoubleComplex *A,  hipDoubleComplex *B, 
        hipDoubleComplex *C, const int n){
    int lda=n,ldb=n,ldc=n;
    const hipDoubleComplex alf = {1.,1.};
    const hipDoubleComplex bet = {0.,0.};
    const hipDoubleComplex *alpha = &alf;
    const hipDoubleComplex *beta = &bet;
    hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, alpha, 
            reinterpret_cast< const hipDoubleComplex*>(A), lda, 
            reinterpret_cast< const hipDoubleComplex*>(B), ldb, beta,
            reinterpret_cast< hipDoubleComplex*>(C), ldc);
}

void gpu_blas_sgemm(hipblasHandle_t handle, const float *A, const float *B, float *C, const int n){
    int lda=n,ldb=n,ldc=n;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, alpha, A, lda, B, ldb, beta, C, ldc);
}
void gpu_blas_dgemm(hipblasHandle_t handle, const double *A, const double *B, double *C, const int n){
    int lda=n,ldb=n,ldc=n;
    const double alf = 1;
    const double bet = 0;
    const double *alpha = &alf;
    const double *beta = &bet;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, alpha, A, lda, B, ldb, beta, C, ldc);
}

/* MAIN functions */

void MatrixMultiplyReal(const int N, const int block_size){
    assert(N%BSIZE == 0);
    fprintf(stderr, "Starting MatrixMultiplyReal...\n");
    // Allocate host memory for matrices A and B
    unsigned int mem_size_A = sizeof(double) * N * N;
    unsigned int mem_size_B = sizeof(double) * N * N;
    unsigned int mem_size_C = sizeof(double) * N * N;
    unsigned int mem_size_fA = sizeof(float) * N * N;
    unsigned int mem_size_fB = sizeof(float) * N * N;
    unsigned int mem_size_fC = sizeof(float) * N * N;

    double *h_A = reinterpret_cast<double *>(malloc(mem_size_A));
    double *h_B = reinterpret_cast<double *>(malloc(mem_size_B));
    double *h_C = reinterpret_cast<double *>(malloc(mem_size_C));
    float *h_fA = reinterpret_cast<float *>(malloc(mem_size_fA));
    float *h_fB = reinterpret_cast<float *>(malloc(mem_size_fB));
    float *h_fC = reinterpret_cast<float *>(malloc(mem_size_fC));

    if (h_A == NULL || h_B == NULL || h_C == NULL
            || h_fA == NULL || h_fB== NULL || h_fC == NULL) {
        fprintf(stderr, "Failed to allocate host matrix A or B or C or fA or fB or fC!\n");
        exit(EXIT_FAILURE);
    }

    // kernel iterations according to matrix size
    int nIter = 1;
    if (N>4000)
        nIter = 150;
    if (N>6000)
        nIter = 100;
    if (N>10000)
        nIter = 50;
    if (N>14000)
        nIter = 10;

    // Allocate device memory
    double *d_A, *d_B, *d_C;
    float *d_fA, *d_fB, *d_fC;

    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_fA), mem_size_A));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_fB), mem_size_B));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_fC), mem_size_C));

    GPU_fill_rand(d_A,N);
    GPU_fill_rand(d_B,N);
    // Setup execution parameters
    const int array = N*N/block_size;
    fillFloatMatrices<BSIZE> <<< array, block_size>>>(d_A, d_fA);
    fillFloatMatrices<BSIZE> <<< array, block_size>>>(d_B, d_fB);
#ifdef DEBUG
    checkCudaErrors(hipMemcpy(h_A, d_A, mem_size_A, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_B, d_B, mem_size_B, hipMemcpyDeviceToHost));
    printf("A\n");
    print_dmatrix(h_A,N,true);
    printf("B\n");
    print_dmatrix(h_B,N,true);

    checkCudaErrors(hipMemcpy(h_fA, d_fA, mem_size_fA, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_fB, d_fB, mem_size_fB, hipMemcpyDeviceToHost));
    printf("fA\n");
    print_fmatrix(h_fA,N,true);
    printf("fB\n");
    print_fmatrix(h_fB,N,true);
#endif

    float double_msecTotal = 0.0f, float_msecTotal = 0.0f;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipEvent_t start;
    checkCudaErrors(hipEventCreate(&start));
    hipEvent_t stop;
    checkCudaErrors(hipEventCreate(&stop));

    hipDeviceSynchronize();
    // Allocate CUDA events that we'll use for timing
    // Record the start event
    checkCudaErrors(hipEventRecord(start, NULL));
    for (int j = 0; j < nIter; j++)
        gpu_blas_dgemm(handle, d_A, d_B, d_C,N);
    // Record the stop event
    checkCudaErrors(hipEventRecord(stop, NULL));
    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&double_msecTotal, start, stop));

    hipDeviceSynchronize();
    checkCudaErrors(hipEventRecord(start, NULL));
    for (int j = 0; j < nIter; j++)
        gpu_blas_sgemm(handle,d_fA,d_fB, d_fC,N);
    checkCudaErrors(hipEventRecord(stop, NULL));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&float_msecTotal, start, stop));

    hipblasDestroy(handle);

    // Copy result from device to host
    checkCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_fC, d_fC, mem_size_fC, hipMemcpyDeviceToHost));

    // Compute and print the performance
    computeAndPrintPerf(N, double_msecTotal, float_msecTotal, nIter);

    // Clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_fA);
    free(h_fB);
    free(h_fC);
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));
    checkCudaErrors(hipFree(d_fA));
    checkCudaErrors(hipFree(d_fB));
    checkCudaErrors(hipFree(d_fC));
}
